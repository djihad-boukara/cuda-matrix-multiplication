#include "hip/hip_runtime.h"
#include "utils.h"
#include "cuda_mat_lib.h"

#define MAT_SIZE 10000

int main(int argc, char const* argv[])
{
	double* A_h, * B_h, * C_d; // matrixes
	double gflops_gpu, gpuTime;
	float elapsed;

	// event variables used to estimate 
	// the execution times on the GPU
	hipEvent_t start_gpu, stop_gpu;
	hipError_t cudaStatus;

	// matrices allocation and filling
	A_h = rnd_flt_matrix(MAT_SIZE, MAT_SIZE);
	B_h = rnd_flt_matrix(MAT_SIZE, MAT_SIZE);
	C_d = zeros_flt_matrix(MAT_SIZE, MAT_SIZE);

	// initiates start and end events
	hipEventCreate(&start_gpu);
	hipEventCreate(&stop_gpu);

	printf("Matrix size; Time; GFlops\n");

	for (int i = 100; i <= MAT_SIZE; i += 100)
	{
		// records the start event
		hipEventRecord(start_gpu, 0);

		// run product on the GPU @see matmatCuda
		cudaStatus = matmatCuda(MAT_SIZE, MAT_SIZE, MAT_SIZE,
					i, i, i, A_h, B_h, C_d);

		// records the end event
		hipEventRecord(stop_gpu, 0);
		hipEventSynchronize(stop_gpu);

		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "matmatCuda failed!");
			return 1;
		}

		// calculates the time elapsed between the 2 events
		hipEventElapsedTime(&elapsed, start_gpu, stop_gpu);

		// turns milliseconds into seconds 
		gpuTime = elapsed / 1000.;

		gflops_gpu = ((2 * pow(i, 3)) / gpuTime) / GIGA;

		printf("%6d; %5.2lf; %5.2lf\n", i, gpuTime, gflops_gpu);
	}

	// destroy cuda events
	hipEventDestroy(start_gpu);
	hipEventDestroy(stop_gpu);

	// free memory
	free(A_h);
	free(B_h);
	free(C_d);

	return 0;
}

